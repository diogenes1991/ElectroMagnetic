#include "hip/hip_runtime.h"
#include <iostream>
#include <sched.h>
#include <pthread.h>
#include <thread>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <unistd.h>
#include <random> 

#include"cloak.h"

#define GRANULARITY 13
#define NDIMENSIONS 8

/// Binary Preprocessor Calculator

#define XOR_0_0 0
#define XOR_0_1 1
#define XOR_1_0 1
#define XOR_1_1 0
#define XOR(X,Y) XOR_##X##_##Y

#define AND_0_0 0
#define AND_0_1 0
#define AND_1_0 0
#define AND_1_1 1
#define AND(X,Y) AND_##X##_##Y

#define REMOVE_PARS(...) EXPAND __VA_ARGS__
#define FIRST(X,...) X
#define TRIM_FIRST(X,...) __VA_ARGS__    

#define BIT_ADD(X,Y,...) (XOR(X,Y)IF(AND(X,Y))(COMMA()AND(X,Y),))

#define MUL(X,Y)*Y
#define POW(X,N) 1 REPEAT(N,MUL,X)

#define BUNCH_SIZE 1<<20
#define CREATE_POTENTIAL(N,J) T A ##J## _ ##N [int(POW(GRANULARITY,NDIMENSIONS-1))];
#define CREATE_COPIES_INDIRECT(J,M) REPEAT(M,CREATE_POTENTIAL,J)
#define CREATE_COPIES(J,M) EVAL(REPEAT(J,CREATE_COPIES_INDIRECT,M))

template <class T>
class ElectroMagneticField{
    public:
        
    T Potential[NDIMENSIONS][GRANULARITY][GRANULARITY];
    T DPotential[NDIMENSIONS][GRANULARITY][GRANULARITY];
            
    ElectroMagneticField();
    
    // The class has several types of grid topologies 
    // the default for now is that the (N-1)-Space part 
    // is a (N-1)-Genus surface. Essentially the grid 
    // has periodic boundary conditions.
    
    void Update_Potential(int topology = 0);
    void Show_Potential(int Component);
    void Show_DPotential(int Component);
    void Show_Field(int mu, int nu);
    T Compute_Field(int mu, int nu, int x, int y);
    T DmuAnu(int mu, int nu, int x, int y);
    T Energy();
    
    
};

template <class T>
ElectroMagneticField<T>::ElectroMagneticField(){
    for (int k=0;k<NDIMENSIONS;k++){
            for(int i=0;i<GRANULARITY;i++){
                for(int j=0;j<GRANULARITY;j++){
                    Potential[k][i][j] = 0.;
                    DPotential[k][i][j] = 0.;
                }
            }
    }
}

template <class T>
void ElectroMagneticField<T>::Update_Potential(int Top){
    
    
    /// Periodic Boundary
    if(Top==0){
        for (int k=0;k<NDIMENSIONS;k++){
            for(int i=0;i<GRANULARITY;i++){
                for(int j=0;j<GRANULARITY;j++){
                
                    /// First we update the Value of the Potentials
                    Potential[k][i][j] += DPotential[k][i][j];
                
                    T val = 0;
                    /// First Component
                    val += (Potential[k][(i+2)%GRANULARITY][j] + Potential[k][(GRANULARITY+i-2)%GRANULARITY][j] - 2*Potential[k][i][j])/4;
                    val += (Potential[k][i][(j+2)%GRANULARITY] + Potential[k][i][(GRANULARITY+j-2)%GRANULARITY] - 2*Potential[k][i][j])/4;
                    DPotential[k][i][j] = val;
                }
            }
        
        }
    }
    
}

template <class T>
void ElectroMagneticField<T>::Show_Potential(int Component){
    
    std::cout << Component << "-component of the potential" <<std::endl;
    for(int i=0;i<GRANULARITY;i++){
            std::cout << "| ";
            for(int j=0;j<GRANULARITY;j++){
                    std::cout << Potential[Component][i][j] << " ";
            }
            std::cout << "|" <<std::endl;
    }
    
    
}

template <class T>
void ElectroMagneticField<T>::Show_DPotential(int Component){
    
    std::cout << Component << " component of the derivative of the potential" <<std::endl;
    for(int i=0;i<GRANULARITY;i++){
            std::cout << "| ";
            for(int j=0;j<GRANULARITY;j++){
                    std::cout << DPotential[Component][i][j] << " ";
            }
            std::cout << "|" <<std::endl;
    }
    
    
}

template <class T>
void ElectroMagneticField<T>::Show_Field(int mu, int nu){
    std::cout << "("<<mu<<","<<nu<<")-component of the field" <<std::endl;
    for(int i=0;i<GRANULARITY;i++){
            std::cout << "| ";
            for(int j=0;j<GRANULARITY;j++){
                std::cout << Compute_Field(mu,nu,i,j) << " ";
           }
            std::cout << "|" <<std::endl;
    }
}

template <class T>
T ElectroMagneticField<T>::DmuAnu(int mu, int nu, int x, int y){
    T rval = 0;
    switch (mu){
        case 0:
            rval = DPotential[nu][x][y];
            break;
        case 1:
            rval = (Potential[nu][(x+1)%GRANULARITY][y]+Potential[nu][(GRANULARITY+x-1)%GRANULARITY][y])/2;
            break;
        case 2:
            rval = (Potential[nu][x][(y+1)%GRANULARITY]+Potential[nu][x][(GRANULARITY+y-1)%GRANULARITY])/2;
            break;
    }
    return rval;
}

template <class T>
T ElectroMagneticField<T>::Compute_Field(int mu, int nu, int x, int y){
    T rval = DmuAnu(mu,nu,x,y)-DmuAnu(nu,mu,x,y);
    return rval;
}

template <class T>
T ElectroMagneticField<T>::Energy(){
    T U = 0;
    for(int i=0;i<GRANULARITY;i++){
            for(int j=0;j<GRANULARITY;j++){
                for(int k = 0;k<NDIMENSIONS;k++){
                    for(int m = k+1;m<NDIMENSIONS;m++){
                        T aux = Compute_Field(k,m,i,j);
                        U += aux*aux/2;
                        }
                    }
           }
    }
    return U;
}

void VECT_TO_FLAT(int* vect_x, int* flat_x){
        *flat_x = 0;
        for(int i=0;i<NDIMENSIONS-1;i++){
            *flat_x += vect_x[i]*pow(GRANULARITY,i);
        }
}

void FLAT_TO_VECT(int* vect_x, int* flat_x){
    int aux = *flat_x;
    for(int i=0;i<NDIMENSIONS-1;i++){
            vect_x[i] = aux%GRANULARITY;
            aux -= aux%GRANULARITY;
            aux /= GRANULARITY;
    }
}

int main(void){
    
    typedef double T;
    CREATE_COPIES(4,1)
     
    for (int i=0;i<pow(GRANULARITY,NDIMENSIONS-1);i++){
        A0_0[i] = i;
        
    }
//     
    std::cout<<"| ";
    for (int i=0;i<pow(GRANULARITY,NDIMENSIONS-1);i++){
        std::cout << A0_0[i] << " ";
        if(i%GRANULARITY == GRANULARITY-1)
            if(i==pow(GRANULARITY,NDIMENSIONS-1)-1)std::cout << "|\n";
            else std::cout << "|\n| ";
        
    }
//     
//     dA0_0[x] ~ SUM(2*NDIM Terms)/(2*NDIM) - A0_0[x] 
//     
//     A0_0[x][y][z] ~ A0_0[x+2][y][z] + A0_0[x-2][y][z] + ...
//     
//     100 ~ x[2] = {15,4}
//     {15,4} ~ 100 
//     
//     INV_MAP(100) = {15,4}
//     A0_0(MAP({15+2,4}) + A0_0({15-2,4})
//     
    
//     ElectroMagneticField<double> E;
//     E.Potential[0][10][10] = 100;
//     for(int i=0;i<1000;i++){
//         E.Update_Potential();
//         if(i%10==0){
//             std::cout<<"U = ("<<i/10<<"%) = "<< E.Energy()<<std::endl;
//         }
//     }
//     
//     E.Show_Potential(0);
//     E.Show_Field(0,1);
//     E.Show_Field(0,2);
//     E.Show_Field(1,2);
    
    int h[NDIMENSIONS-1];
    int g = 170;
    #if 3>110
    CREATE_COPIES(3,5)
    #endif
    
    
    FLAT_TO_VECT(h,&g);
    VECT_TO_FLAT(h,&g);
    std::cout<<"("<<h[0]<<","<<h[1]<<","<<h[2]<<")"<<std::endl;
    std::cout<<"("<<g<<")"<<std::endl;
    
}
