
#include <hip/hip_runtime.h>
#include <iostream>
#include <sched.h>
#include <pthread.h>
#include <thread>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <unistd.h>
#include <random> 

#define GRANULARITY 10
#define NDIMENSIONS 3

template <class T>
class ElectroMagneticField{
    public:
    
    T Potential[NDIMENSIONS][GRANULARITY][GRANULARITY];
    T DPotential[NDIMENSIONS][GRANULARITY][GRANULARITY];
        
    ElectroMagneticField();
    
    // The class has several types of grid topologies 
    // the default for now is that the (N-1)-Space part 
    // is a (N-1)-Genus surface. Essentially the grid 
    // has periodic boundary conditions.
    
    void Update_Potential(int topology = 0);
    void Show_Potential(int Component);
    void Show_DPotential(int Component);
    void Show_Field(int mu, int nu);
    T Compute_Field(int mu, int nu, int x, int y);
    T DmuAnu(int mu, int nu, int x, int y);
    T Energy();
    
    
};

template <class T>
ElectroMagneticField<T>::ElectroMagneticField(){
    for (int k=0;k<NDIMENSIONS;k++){
            for(int i=0;i<GRANULARITY;i++){
                for(int j=0;j<GRANULARITY;j++){
                    Potential[k][i][j] = 0.;
                    DPotential[k][i][j] = 0.;
                }
            }
    }
}

template <class T>
void ElectroMagneticField<T>::Update_Potential(int Top){
    
    
    /// Periodic Boundary
    if(Top==0){
        for (int k=0;k<NDIMENSIONS;k++){
            for(int i=0;i<GRANULARITY;i++){
                for(int j=0;j<GRANULARITY;j++){
                
                    /// First we update the Value of the Potentials
                    Potential[k][i][j] += DPotential[k][i][j];
                
                    T val = 0;
                    /// First Component
                    val += (Potential[k][(i+2)%GRANULARITY][j] + Potential[k][(GRANULARITY+i-2)%GRANULARITY][j] - 2*Potential[k][i][j])/4;
                    val += (Potential[k][i][(j+2)%GRANULARITY] + Potential[k][i][(GRANULARITY+j-2)%GRANULARITY] - 2*Potential[k][i][j])/4;
                    DPotential[k][i][j] = val;
                }
            }
        
        }
    }
    
}

template <class T>
void ElectroMagneticField<T>::Show_Potential(int Component){
    
    std::cout << Component << "-component of the potential" <<std::endl;
    for(int i=0;i<GRANULARITY;i++){
            std::cout << "| ";
            for(int j=0;j<GRANULARITY;j++){
                    std::cout << Potential[Component][i][j] << " ";
            }
            std::cout << "|" <<std::endl;
    }
    
    
}

template <class T>
void ElectroMagneticField<T>::Show_DPotential(int Component){
    
    std::cout << Component << " component of the derivative of the potential" <<std::endl;
    for(int i=0;i<GRANULARITY;i++){
            std::cout << "| ";
            for(int j=0;j<GRANULARITY;j++){
                    std::cout << DPotential[Component][i][j] << " ";
            }
            std::cout << "|" <<std::endl;
    }
    
    
}

template <class T>
void ElectroMagneticField<T>::Show_Field(int mu, int nu){
    std::cout << "("<<mu<<","<<nu<<")-component of the derivative of the potential" <<std::endl;
    for(int i=0;i<GRANULARITY;i++){
            std::cout << "| ";
            for(int j=0;j<GRANULARITY;j++){
                std::cout << Compute_Field(mu,nu,i,j) << " ";
           }
            std::cout << "|" <<std::endl;
    }
}

template <class T>
T ElectroMagneticField<T>::DmuAnu(int mu, int nu, int x, int y){
    T rval = 0;
    switch (mu){
        case 0:
            rval = DPotential[nu][x][y];
            break;
        case 1:
            rval = (Potential[nu][x+1][y]+Potential[nu][x-1][y])/2;
            break;
        case 2:
            rval = (Potential[nu][x][y+1]+Potential[nu][x][y-1])/2;
            break;
    }
    return rval;
}

template <class T>
T ElectroMagneticField<T>::Compute_Field(int mu, int nu, int x, int y){
    T rval = DmuAnu(mu,nu,x,y)-DmuAnu(nu,mu,x,y);
    return rval;
}

template <class T>
T ElectroMagneticField<T>::Energy(){
    T U = 0;
    for(int i=0;i<GRANULARITY;i++){
            for(int j=0;j<GRANULARITY;j++){
                for(int k = 0;k<NDIMENSIONS;k++){
                    for(int m = k+1;m<NDIMENSIONS;m++){
                        T aux = Compute_Field(k,m,i,j);
                        U += aux*aux/2;
                        }
                    }
           }
    }
    return U;
}

int main(void){

    std::cout.precision(4);
    ElectroMagneticField<double> E;
    E.Potential[0][5][0] = 10;
//     E.Potential[0][6][1] = 13;
//     E.Potential[0][5][2] = 11;
//     E.Potential[0][6][3] = 9;
    E.Show_Potential(0);
    std::cout << "Energy = "<<E.Energy()<<std::endl;
    for(int i=0;i<50;i++){
    E.Update_Potential();
//     E.Show_Potential(0);
    }
    E.Show_Potential(0);
    E.Show_Potential(1);
    E.Show_Potential(2);
    std::cout << "Energy = "<<E.Energy()<<std::endl;
    
    
}
